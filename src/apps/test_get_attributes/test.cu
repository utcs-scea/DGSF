
#include <hip/hip_runtime.h>
#include <stdio.h>
 
#define CHECK(call) \
{ \
 const hipError_t error = call; \
 if (error != hipSuccess) \
 { \
     printf("Error: %s:%d, ", __FILE__, __LINE__); \
     printf("code:%d, reason: %s\n", error, hipGetErrorString(error)); \
     exit(1); \
 } \
}

__global__ void dummyKernel(float* data) {
    return;
}

int main(int argc, char* argv[]) {

    float *gpu1data;

    // Enable peer access
    hipSetDevice(1);
    hipMalloc(&gpu1data, 1000);

    printf("Enabling peer access on device 0 to device 1\n");

    hipSetDevice(0);
    CHECK(hipDeviceEnablePeerAccess(1,0));
    hipPointerAttribute_t attr;

    for (int i = 0; i < 3; i++) {
        hipMemset(gpu1data, 1, 1000);
        hipError_t status = hipPointerGetAttributes(&attr, gpu1data);
        printf("\ncudaPointerGetAttr returned: %d\n", status);
        printf("\n[Attributes]\n");
        printf("device: %d\n", attr.device);
        printf("devicePointer: %p\n", attr.devicePointer);
        printf("hostPointer: %p\n", attr.hostPointer);
    }

    printf("\n*** Done ***\n");
}
