#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdint.h>

/*
__global__ void sum(int* dest, size_t n, int val) {
    size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < n) {
       dest[tid] += val;
    }
}
*/

__global__ void sum() {
    size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < 10) {

    }
}

int main(int argc, char* argv[]) {
    int *d_a;
    int *d_b;
    hipError_t err;
    err = hipMalloc((void**)&d_a, 16*1024*1024);
    printf("16MB hipMalloc returned: %d: %s\n", err, hipGetErrorString(err));
    printf("ptr is %p\n", d_a);

    err = hipMemset((void*)d_a, 0, 4096 * sizeof(int));
    printf("hipMemset returned: %d: %s\n", err, hipGetErrorString(err));
    
    //sum<<<1, 64>>>(d_a, 64, 1);
    sum<<<1, 64>>>();
    hipDeviceSynchronize();

    char buf[4096 * sizeof(int)];
    err = hipMemcpy((void*)buf, d_a, 4096 * sizeof(int), hipMemcpyDeviceToHost);
    printf("hipMemcpy returned: %d: %s\n", err, hipGetErrorString(err));

    printf("first 10 ints after 0+1: \n");
    for (int i = 0 ; i < 10 ; i++)
        printf("%#02x ", ((int*)buf)[i]);
    
    err = hipMalloc((void**)&d_b, 16*1024*1024);
    printf("\n2nd 16MB hipMalloc returned: %d: %s\n", err, hipGetErrorString(err));

    err = hipMemset((void*)d_b, 0, 64 * sizeof(int));
    printf("hipMemset returned: %d: %s\n", err, hipGetErrorString(err));

    sum<<<1, 64>>>();
    hipDeviceSynchronize();

    sum<<<1, 64>>>();
    hipDeviceSynchronize();

    err = hipMemcpy((void*)buf, d_b, 64 * sizeof(int), hipMemcpyDeviceToHost);
    printf("hipMemcpy returned: %d: %s\n", err, hipGetErrorString(err));

    printf("first 10 ints of 2nd array (should be 1): \n");
    for (int i = 0 ; i < 10 ; i++)
        printf("%#02x ", ((int*)buf)[i]);

    hipDeviceSynchronize();
    return 0;
}
