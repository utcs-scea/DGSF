#include <hip/hip_runtime.h>

#include <iostream>



int main() {
    hipError_t r;


    hipSetDevice(0);
    hipFree(0);

    hipSetDevice(1);
    hipFree(0);

    hipSetDevice(0);

    //cuInit(0);

    // Create module from binary file
    hipModule_t cuModule;
    r = hipModuleLoad(&cuModule, "minimal.ptx");
    std::cout << "return of cuModuleLoad: " << r << std::endl;

    hipFunction_t sum;
    r = hipModuleGetFunction(&sum, cuModule, "_Z3sumv");
    std::cout << "return of cuModuleGetFunction: " << r << std::endl;

    r = hipModuleLaunchKernel(sum, 1, 1, 1, 4, 1, 1, 0, 0, 0, 0);
    std::cout << "return of cuLaunchKernel: " << r << std::endl;

    std::cout << "!!!  switching context to device 1!" << std::endl;

    hipSetDevice(1);

    r = hipModuleLoad(&cuModule, "minimal.ptx");
    std::cout << "return of cuModuleLoad: " << r << std::endl;

    r = hipModuleGetFunction(&sum, cuModule, "_Z3sumv");
    std::cout << "return of cuModuleGetFunction: " << r << std::endl;

    r = hipModuleLaunchKernel(sum, 1, 1, 1, 4, 1, 1, 0, 0, 0, 0);
    std::cout << "return of cuLaunchKernel: " << r << std::endl;


    return 0;


}
