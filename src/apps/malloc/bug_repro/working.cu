#include <hip/hip_runtime.h>
#include <iostream>



int main() {
    hipError_t r;

    hipInit(0);

    std::cout << "!!!  creating context on device 0!" << std::endl;

    // Get handle for device 0
    hipDevice_t cuDevice;
    r = hipDeviceGet(&cuDevice, 0);
    std::cout << "return of hipDeviceGet: " << r << std::endl;

    // Create context
    hipCtx_t cuContext0, cuContext1;
    r = hipCtxCreate(&cuContext0, 0, cuDevice);
    std::cout << "return of hipCtxCreate: " << r << std::endl;

    // Create module from binary file
    hipModule_t cuModule;
    r = hipModuleLoad(&cuModule, "minimal.ptx");
    std::cout << "return of hipModuleLoad: " << r << std::endl;

    hipFunction_t sum0, sum1;
    r = hipModuleGetFunction(&sum0, cuModule, "_Z3sumv");
    std::cout << "return of hipModuleGetFunction: " << r << std::endl;

    printf("sum0: %p\n", sum0);

    std::cout << "loading on context 1 " << std::endl;
    r = hipDeviceGet(&cuDevice, 1);
    std::cout << "return of hipDeviceGet: " << r << std::endl;
    r = hipCtxCreate(&cuContext1, 0, cuDevice);
    std::cout << "return of hipCtxCreate: " << r << std::endl;
    r = hipModuleLoad(&cuModule, "minimal.ptx");
    std::cout << "return of hipModuleLoad: " << r << std::endl;
    r = hipModuleGetFunction(&sum1, cuModule, "_Z3sumv");
    std::cout << "return of hipModuleGetFunction: " << r << std::endl;

    printf("sum1: %p\n", sum1);

    r = hipCtxSetCurrent(cuContext0);
    std::cout << "return of hipCtxSetCurrent 0: " << r << std::endl;

    r = hipModuleLaunchKernel(sum0, 1, 1, 1, 4, 1, 1, 0, 0, 0, 0);
    std::cout << "return of hipModuleLaunchKernel: " << r << std::endl;

    std::cout << "!!!  switching context to device 1!" << std::endl;

    r = hipCtxSetCurrent(cuContext1);
    std::cout << "return of hipCtxSetCurrent 1: " << r << std::endl;

    r = hipModuleLaunchKernel(sum1, 1, 1, 1, 4, 1, 1, 0, 0, 0, 0);
    std::cout << "return of hipModuleLaunchKernel: " << r << std::endl;


    return 0;


}
