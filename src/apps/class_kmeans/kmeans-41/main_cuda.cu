#include <iostream>
#include <vector>
#include <limits>
#include <chrono>
#include "loader.h"
#include "argparse.h"
#include "rng.h"
#include "hip/hip_runtime.h"
#include "kmeans_kernel.cuh"

int main(int argc, char** argv) {
    hipEvent_t start, end, start_exe_only, end_exe_only;
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventCreate(&start_exe_only);
    hipEventCreate(&end_exe_only);
    hipEventRecord(start);
    
    options_t opts;
    get_opts(argc, argv, &opts);
    loader file_loader(opts.dims);
    
    int dims = opts.dims;
    int ncentroids = opts.num_cluster;
    double* points = nullptr;
    double* centroids = nullptr;
    int npoints;
    file_loader.load_as_pointer(opts.inputfilename, &points, npoints);
    int points_size = npoints * dims;
    int centroids_size = ncentroids * dims;
    int cross_size = npoints * ncentroids;
    
    centroids = (double*)malloc(centroids_size * sizeof(double));
    rng randomizer(opts.seed);
    for (int i = 0; i < opts.num_cluster; i++) {
        int idx = randomizer.kmeans_rand() % npoints;
        for (int j = 0; j < dims; j++) {
            centroids[i * dims + j] = points[idx * dims + j];
        }
    }
    
    double *d_points, *d_centroids, *d_distances, *d_new_centroids, *d_convergence_distances;
    int *d_minimum_centroid_ids, *d_centroid_counts, *d_cnv;
    hipMalloc((void**)&d_points, points_size * sizeof(double));
    hipMalloc((void**)&d_centroids, centroids_size * sizeof(double));
    hipMalloc((void**)&d_distances, cross_size * sizeof(double));
    hipMalloc((void**)&d_new_centroids, centroids_size * sizeof(double));
    hipMalloc((void**)&d_minimum_centroid_ids, npoints * sizeof(int));
    hipMalloc((void**)&d_centroid_counts, ncentroids * sizeof(int));
    hipMalloc((void**)&d_convergence_distances, ncentroids * sizeof(double));
    hipMalloc((void**)&d_cnv, sizeof(int));
    
    hipMemcpy(d_points, points, points_size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_centroids, centroids, centroids_size * sizeof(double), hipMemcpyHostToDevice);
    
    bool is_conv = false;
    int iter = 0;
    int cnv[1];
    
    hipEventRecord(start_exe_only);
    
    
    while (!is_conv && iter < opts.max_num_iter) {
        hipMemset(d_centroid_counts, 0, ncentroids * sizeof(int));
        hipMemset(d_cnv, 0, sizeof(int));
        reset_zero<<<ncentroids, dims>>>(d_new_centroids, centroids_size);
        
        if (opts.cuda_shared) {
            nearest_centroid_shared<<<npoints, ncentroids>>>(d_points, d_centroids, dims, npoints, ncentroids, points_size, centroids_size, cross_size, d_minimum_centroid_ids);
        } else {
            dot<<<npoints, ncentroids>>>(d_points, d_centroids, dims, npoints, ncentroids, points_size, centroids_size, cross_size, d_distances);
            nearest_centroid<<<npoints, 1>>>(d_distances, npoints, ncentroids, cross_size, d_minimum_centroid_ids);            
        }

        count_centroid_id<<<npoints, 1>>>(d_minimum_centroid_ids, npoints, ncentroids, d_centroid_counts);        
        sum_new_centroid_values<<<npoints, dims>>>(d_points, d_minimum_centroid_ids, dims, npoints, points_size, centroids_size, d_new_centroids);
        avg_new_centroid_values<<<ncentroids, dims>>>(d_new_centroids, d_centroid_counts, dims, ncentroids, centroids_size); 
        new_centroid_movement_squared<<<ncentroids, dims>>>(d_centroids, d_new_centroids, dims, ncentroids, centroids_size, d_convergence_distances);
        sqrt_kernel<<<1, ncentroids>>>(d_convergence_distances, ncentroids);
        is_convergent<<<1, ncentroids>>>(d_convergence_distances, opts.threshold, ncentroids, d_cnv);
        hipMemcpy(cnv, d_cnv, sizeof(int), hipMemcpyDeviceToHost);
        is_conv = (cnv[0] == ncentroids);
        iter++;
        hipMemcpy(d_centroids, d_new_centroids, centroids_size * sizeof(double), hipMemcpyDeviceToDevice);
    }
    
    hipEventRecord(end_exe_only);
    hipEventSynchronize(end_exe_only);
    hipEventRecord(end);
    
    float diff = 0;
    hipEventElapsedTime(&diff, start_exe_only, end_exe_only);
    
    printf("%d,%f\n", iter, diff);
    if (opts.output_centroids) {
        hipMemcpy(centroids, d_centroids, centroids_size * sizeof(double), hipMemcpyDeviceToHost);
        for (int i = 0; i < ncentroids; i++) {
            printf("%d ", i);
            for (int j = 0; j < dims; j++) {
                printf("%lf ", centroids[j + dims * i]);
            }
            printf("\n");
        }
    } else {
        int clusters[npoints];
        hipMemcpy(clusters, d_minimum_centroid_ids, npoints * sizeof(int), hipMemcpyDeviceToHost);
        printf("clusters:");
        for (int i = 0; i < npoints; i++) {
            printf(" %d", clusters[i]);
        }
    }
    
    if (opts.print_e2e) {
        hipEventElapsedTime(&diff, start, end);
        printf("%f\n", diff);
    }

    free(points);
    free(centroids);
    hipFree(d_points);
    hipFree(d_centroids);
    hipFree(d_distances);
    hipFree(d_new_centroids);
    hipFree(d_minimum_centroid_ids);
    hipFree(d_centroid_counts);
    hipFree(d_convergence_distances);

    return 0;
}